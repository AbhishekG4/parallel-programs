// Engineer: Abhishek Gautam
// Last Updated: 02-22-2025

// Problem/Functionality: This is a Kernel to perform a reduction on an input
//                        array. In this case we are simply going to sum all
//                        elements of the array.

// High-level Approach:
// 1) The array is to be divided into segments processed by thread blocks of
//    a particular maximum size.
// 2) A tree based approach is used within each thread block.
//     2.1) Each block has as many elements to work with as threads.
//     2.2) A loop is run in which in each cycle the threads reduce the
//          block data to half its size until only 1 element is left which
//          is the result of the reduction on the block
//     2.3) The block's result is stored at its block index in the input
//          array.
// 3) The problem size has now shrunk from N to no. of blocks launced. This
//    process along with the kernel launch is repeated in a loop on the
//    results of the thread blocks until 1 thread block is left which
//    eveluates to one result which is the final result.
// 4) As an optimization, each thread brings in two values from the input
//    array into shared memory. Thus, each block processes twice the size
//    of input elements. This helps distribute work among the threads more
//    evenly threads start to become idle as the process continues.
// 5) As an optimization, the last warp(32 threads) worth of work is
//    unrolled. This helps avoid instruction overhead and reduces useless
//    work in the other warps.

// =============================================================================


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N_ (64 * 1024 * 1024)
#define MAX_THREADS 512

typedef double dtype;

int nxtPow2(int x) {  // Obtains the next greatest power of 2 if not already a
                      // power of 2
  x--;
  x |= x >> 1;
  x |= x >> 2;
  x |= x >> 4;
  x |= x >> 8;
  x |= x >> 16;
  return ++x;
}
void GetNumBlocksAndThreads(unsigned int n, int& blocks,
                            int& threads) {  // Assuming first add. Each thread
                                             // brings in two values from mem
  threads = (n > MAX_THREADS * 2) ? MAX_THREADS : nxtPow2((n + 1) / 2);
  blocks = (n + threads * 2 - 1) / (threads * 2);
}

__device__ void WarpUnroll(volatile dtype* blk_mem, unsigned int tid) {
  blk_mem[tid] += blk_mem[tid + 32];
  blk_mem[tid] += blk_mem[tid + 16];
  blk_mem[tid] += blk_mem[tid + 8];
  blk_mem[tid] += blk_mem[tid + 4];
  blk_mem[tid] += blk_mem[tid + 2];
  blk_mem[tid] += blk_mem[tid + 1];
}
__global__ void ReductionKernel(dtype* i_data, unsigned int n, unsigned int N) {
  __shared__ dtype blk_mem[MAX_THREADS];
  unsigned int bid = blockIdx.y * gridDim.x + blockIdx.x;
  unsigned int idx = bid * 2 * blockDim.x + threadIdx.x;

  // bringing in main memory to block memory
  if (idx >= N)
    blk_mem[threadIdx.x] = 0.0;
  else if (idx + blockDim.x >= N)
    blk_mem[threadIdx.x] = i_data[idx];
  else
    blk_mem[threadIdx.x] =
        i_data[idx] + i_data[idx + blockDim.x];  // Each thread brings in two
                                                 // elemnts by adding them first
  __syncthreads();

  // Reducing block
  for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
    if (threadIdx.x < s) blk_mem[threadIdx.x] += blk_mem[threadIdx.x + s];
    __syncthreads();
  }
  if (threadIdx.x < 32) WarpUnroll(blk_mem, threadIdx.x);

  // writing back to main memory
  if (threadIdx.x == 0) i_data[bid] = blk_mem[0];
}

dtype Reduction(dtype* i_data_h, unsigned int N) {
  dtype* i_data_d;
  // Move data to GPU
  hipMalloc(&i_data_d, sizeof(dtype) * N);
  hipMemcpy(i_data_d, i_data_h, sizeof(dtype) * N, hipMemcpyHostToDevice);
  // Reduce
  unsigned int n = N;
  int threads = 0;
  int blocks = 0;
  while (n > 1) {
    GetNumBlocksAndThreads(n, blocks, threads);
    // Kernel launch
    dim3 grid_block(16, (blocks + 16 - 1) / 16);
    dim3 thread_block(threads, 1);
    ReductionKernel<<<grid_block, thread_block>>>(i_data_d, n, N);
    n = blocks;  // new problem size
  }
  // Move result back
  hipMemcpy(i_data_h, i_data_d, sizeof(dtype) * 1, hipMemcpyDeviceToHost);
  // Free
  hipFree(i_data_d);

  return i_data_h[0];
}

dtype CPUReduction(dtype* i_data, unsigned int N) {
  dtype sum = 0;
  for (int i = 0; i < N; i++) {
    sum += i_data[i];
  }
  return sum;
}

int main(int argc, char** argv) {
  unsigned int N;
  dtype* i_data_h;

  // Procure arguments from user
  if (argc > 1) {
    N = (unsigned int)atoi(argv[1]);
    printf("N = %u\n", N);
  } else {
    N = N_;
    printf("N = %u\n", N);
  }

  // Prepare random input
  i_data_h = (dtype*)malloc(sizeof(dtype) * N);
  if (!i_data_h) printf("--Malloc failed for i_data_h--\n");
  srand48(21);
  for (int i = 0; i < N; i++) i_data_h[i] = drand48() / 100000;

  // Reduce
  dtype ground_truth = CPUReduction(i_data_h, N);
  dtype device_result = Reduction(i_data_h, N);
  // Verify
  printf("device result = %f\nground  truth = %f\n", device_result,
         ground_truth);
  // Free
  free(i_data_h);
}